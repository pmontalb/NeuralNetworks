#include "hip/hip_runtime.h"
#include <ObjectiveFunctions.cuh>
#include <CubWrappers.cuh>
#include <CuBlasWrappers.cuh>
#include <MemoryManager.cuh>
#include <BufferInitializer.cuh>

template <typename T>
DEVICE T __SigmoidWorker__(const T x)
{
	return static_cast<T>(1.0) / (static_cast<T>(1.0) + exp(-x));
}

template <typename T>
DEVICE T __HyperbolicTangentWorker__(const T x)
{
	return static_cast<T>(2.0) * __SigmoidWorker__<T>(static_cast<T>(2.0) * x) - static_cast<T>(1.0);
}

template <typename T>
DEVICE T __InverseSquareRootLinearUnitDenominatorWorker__(const T x)
{
	return static_cast<T>(1.0) / (static_cast<T>(1.0) + x * x);
}

template <typename T>
DEVICE T __ExponentialLinearUnitPrimeWorker__(const T x)
{
	return exp(x);
}

template <typename T>
DEVICE T __BentIdentityPrimeWorker__(const T x)
{
	return sqrt(x * x + static_cast<T>(1.0));
}

template <typename T>
DEVICE T __CrossEntropyWorker__(const T x, const T y)
{
	return 	x * log(y);
}

template <typename T>
GLOBAL void __Sigmoid__(T* RESTRICT z, const T* RESTRICT x, const unsigned sz)
{
	CUDA_FUNCTION_PROLOGUE;
	
	CUDA_FOR_LOOP_PROLOGUE
		z[i] = __SigmoidWorker__<T>(x[i]);
	CUDA_FOR_LOOP_EPILOGUE
}

template <typename T>
GLOBAL void __SigmoidPrime__(T* RESTRICT z, const T* RESTRICT x, const unsigned sz)
{
	CUDA_FUNCTION_PROLOGUE;
	
	CUDA_FOR_LOOP_PROLOGUE
		const T sigmoidZ = __SigmoidWorker__<T>(x[i]);
		z[i] = sigmoidZ * (static_cast<T>(1.0) - sigmoidZ);
	CUDA_FOR_LOOP_EPILOGUE
}

template <typename T>
GLOBAL void __HyperbolicTangent__(T* RESTRICT z, const T* RESTRICT x, const unsigned sz)
{
	CUDA_FUNCTION_PROLOGUE;
	
	CUDA_FOR_LOOP_PROLOGUE
		z[i] = __HyperbolicTangentWorker__<T>(x[i]);
	CUDA_FOR_LOOP_EPILOGUE
}

template <typename T>
GLOBAL void __HyperbolicTangentPrime__(T* RESTRICT z, const T* RESTRICT x, const unsigned sz)
{
	CUDA_FUNCTION_PROLOGUE;
	
	CUDA_FOR_LOOP_PROLOGUE
		const T tanhZ = __HyperbolicTangentWorker__<T>(x[i]);
		z[i] = static_cast<T>(1.0) - tanhZ * tanhZ;
	CUDA_FOR_LOOP_EPILOGUE
}

template <typename T>
GLOBAL void __RectifiedLinearUnit__(T* RESTRICT z, const T* RESTRICT x, const T alpha, const unsigned sz)
{
	CUDA_FUNCTION_PROLOGUE;
	
	CUDA_FOR_LOOP_PROLOGUE
		if (x[i] <= static_cast<T>(0.0))
			z[i] = alpha * x[i];
		else
			z[i] = x[i];
	CUDA_FOR_LOOP_EPILOGUE
}

template <typename T>
GLOBAL void __RectifiedLinearUnitPrime__(T* RESTRICT z, const T* RESTRICT x, const T alpha, const unsigned sz)
{
	CUDA_FUNCTION_PROLOGUE;
	
	CUDA_FOR_LOOP_PROLOGUE
		if (x[i] <= static_cast<T>(0.0))
			z[i] = alpha;
		else
			z[i] = 1.0;
	CUDA_FOR_LOOP_EPILOGUE
}

template <typename T>
GLOBAL void __InverseSquareRootLinearUnit__(T* RESTRICT z, const T* RESTRICT x, const unsigned sz)
{
	CUDA_FUNCTION_PROLOGUE;
	
	CUDA_FOR_LOOP_PROLOGUE
		if (x[i] <= static_cast<T>(0.0))
			z[i] = x[i] * __InverseSquareRootLinearUnitDenominatorWorker__(x[i]);
		else
			z[i] = x[i];
	CUDA_FOR_LOOP_EPILOGUE
}

template <typename T>
GLOBAL void __InverseSquareRootLinearUnitPrime__(T* RESTRICT z, const T* RESTRICT x, const unsigned sz)
{
	CUDA_FUNCTION_PROLOGUE;
	
	CUDA_FOR_LOOP_PROLOGUE
		if (x[i] <= static_cast<T>(0.0))
		{
			const double factor = __InverseSquareRootLinearUnitDenominatorWorker__(x[i]);
			z[i] = factor * factor * factor;
		}
		else
			z[i] = 1.0;
	CUDA_FOR_LOOP_EPILOGUE
}

template <typename T>
GLOBAL void __ExponentialLinearUnit__(T* RESTRICT z, const T* RESTRICT x, const unsigned sz)
{
	CUDA_FUNCTION_PROLOGUE;
	
	CUDA_FOR_LOOP_PROLOGUE
		if (x[i] <= static_cast<T>(0.0))
			z[i] = __ExponentialLinearUnitPrimeWorker__(x[i]) - static_cast<T>(1.0);
		else
			z[i] = x[i];
	CUDA_FOR_LOOP_EPILOGUE
}

template <typename T>
GLOBAL void __ExponentialLinearUnitPrime__(T* RESTRICT z, const T* RESTRICT x, const unsigned sz)
{
	CUDA_FUNCTION_PROLOGUE;
	
	CUDA_FOR_LOOP_PROLOGUE
		if (x[i] <= static_cast<T>(0.0))
			z[i] = __ExponentialLinearUnitPrimeWorker__(x[i]);
		else
			z[i] = x[i];
	CUDA_FOR_LOOP_EPILOGUE
}

template <typename T>
GLOBAL void __BentIdentity__(T* RESTRICT z, const T* RESTRICT x, const unsigned sz)
{
	CUDA_FUNCTION_PROLOGUE;
	
	CUDA_FOR_LOOP_PROLOGUE
		const T sqrtX2 = __BentIdentityPrimeWorker__(x[i]);
		z[i] = x[i] + static_cast<T>(0.5) * (sqrtX2 - static_cast<T>(1.0));
	CUDA_FOR_LOOP_EPILOGUE
}

template <typename T>
GLOBAL void __BentIdentityPrime__(T* RESTRICT z, const T* RESTRICT x, const unsigned sz)
{
	CUDA_FUNCTION_PROLOGUE;
	
	CUDA_FOR_LOOP_PROLOGUE
		const T sqrtX2 = __BentIdentityPrimeWorker__(x[i]);
		z[i] = static_cast<T>(1.0) + static_cast<T>(0.5) * x[i] / sqrtX2;
	CUDA_FOR_LOOP_EPILOGUE
}

template <typename T>
GLOBAL void __SoftMax__(T* RESTRICT z, const T* RESTRICT x, const unsigned sz)
{
	CUDA_FUNCTION_PROLOGUE;
	
	CUDA_FOR_LOOP_PROLOGUE
		z[i] = exp(x[i]);  // normalised later on!
	CUDA_FOR_LOOP_EPILOGUE
}

template <typename T>
GLOBAL void __CrossEntropyCostFunctionSigmoid__(T* RESTRICT x, const T* RESTRICT y, const unsigned sz)
{
	CUDA_FUNCTION_PROLOGUE;
	
	CUDA_FOR_LOOP_PROLOGUE
		const T crossEntropy = -__CrossEntropyWorker__(y[i], x[i]) - __CrossEntropyWorker__(1.0 - y[i], 1.0 - x[i]);
		if (!isfinite(crossEntropy))
			x[i] = 0.0;
		else
			x[i] = crossEntropy;
	CUDA_FOR_LOOP_EPILOGUE
}

template <typename T>
GLOBAL void __CrossEntropyCostFunctionSoftMax__(T* RESTRICT x, const T* RESTRICT y, const unsigned sz)
{
	CUDA_FUNCTION_PROLOGUE;
	
	CUDA_FOR_LOOP_PROLOGUE
		const T crossEntropy = -__CrossEntropyWorker__(y[i], x[i]);
		if (!isfinite(crossEntropy))
			x[i] = 0.0;
		else
			x[i] = crossEntropy;
	CUDA_FOR_LOOP_EPILOGUE
}


static inline int RectifiedLinearUnitWorker(MemoryBuffer& z, const MemoryBuffer& x, const double alpha)
{
	switch (z.mathDomain)
	{
		case MathDomain::Float:
			CUDA_CALL_SINGLE(__RectifiedLinearUnit__<float>, (float*)z.pointer, (float*)x.pointer, (float)alpha, z.size);
			break;
		case MathDomain::Double:
			CUDA_CALL_DOUBLE(__RectifiedLinearUnit__<double>, (double*)z.pointer, (double*)x.pointer, (double)alpha, z.size);
			break;
		default:
			return CudaKernelException::_NotImplementedException;
	}
	return hipGetLastError();
}

static inline int RectifiedLinearUnitPrimeWorker(MemoryBuffer& z, const MemoryBuffer& x, const double alpha)
{
	switch (z.mathDomain)
	{
		case MathDomain::Float:
			CUDA_CALL_SINGLE(__RectifiedLinearUnitPrime__<float>, (float*)z.pointer, (float*)x.pointer, (float)alpha, z.size);
			break;
		case MathDomain::Double:
			CUDA_CALL_DOUBLE(__RectifiedLinearUnitPrime__<double>, (double*)z.pointer, (double*)x.pointer, (double)alpha, z.size);
			break;
		default:
			return CudaKernelException::_NotImplementedException;
	}
	return hipGetLastError();
}

EXTERN_C
{
	EXPORT int _Sigmoid(MemoryBuffer& z, const MemoryBuffer& x)
	{
		switch (z.mathDomain)
		{
			case MathDomain::Float:
				CUDA_CALL_SINGLE(__Sigmoid__<float>, (float*)z.pointer, (float*)x.pointer, z.size);
				break;
			case MathDomain::Double:
				CUDA_CALL_DOUBLE(__Sigmoid__<double>, (double*)z.pointer, (double*)x.pointer, z.size);
				break;
			default:
				return CudaKernelException::_NotImplementedException;
		}
		return hipGetLastError();
	}
	
	EXPORT int _SigmoidPrime(MemoryBuffer& z, const MemoryBuffer& x)
	{
		switch (z.mathDomain)
		{
			case MathDomain::Float:
				CUDA_CALL_SINGLE(__SigmoidPrime__<float>, (float*) z.pointer, (float*) x.pointer, z.size);
				break;
			case MathDomain::Double:
				CUDA_CALL_DOUBLE(__SigmoidPrime__<double>, (double*) z.pointer, (double*) x.pointer, z.size);
				break;
			default:
				return CudaKernelException::_NotImplementedException;
		}
		return hipGetLastError();
	}

	EXPORT int _HyperbolicTangent(MemoryBuffer& z, const MemoryBuffer& x)
	{
		switch (z.mathDomain)
		{
			case MathDomain::Float:
				CUDA_CALL_SINGLE(__HyperbolicTangent__<float>, (float*)z.pointer, (float*)x.pointer, z.size);
				break;
			case MathDomain::Double:
				CUDA_CALL_DOUBLE(__HyperbolicTangent__<double>, (double*)z.pointer, (double*)x.pointer, z.size);
				break;
			default:
				return CudaKernelException::_NotImplementedException;
		}
		return hipGetLastError();
	}
	
	EXPORT int _HyperbolicTangentPrime(MemoryBuffer& z, const MemoryBuffer& x)
	{
		switch (z.mathDomain)
		{
			case MathDomain::Float:
				CUDA_CALL_SINGLE(__HyperbolicTangentPrime__<float>, (float*) z.pointer, (float*) x.pointer, z.size);
				break;
			case MathDomain::Double:
				CUDA_CALL_DOUBLE(__HyperbolicTangentPrime__<double>, (double*) z.pointer, (double*) x.pointer, z.size);
				break;
			default:
				return CudaKernelException::_NotImplementedException;
		}
		return hipGetLastError();
	}

	EXPORT int _RectifiedLinearUnit(MemoryBuffer& z, const MemoryBuffer& x)
	{
		return RectifiedLinearUnitWorker(z, x, 0.0);
	}

	EXPORT int _RectifiedLinearUnitPrime(MemoryBuffer& z, const MemoryBuffer& x)
	{
		return RectifiedLinearUnitPrimeWorker(z, x, 0.0);
	}

	EXPORT int _LeakyRectifiedLinearUnit(MemoryBuffer& z, const MemoryBuffer& x)
	{
		return RectifiedLinearUnitWorker(z, x, 0.01);
	}
	
	EXPORT int _LeakyRectifiedLinearUnitPrime(MemoryBuffer& z, const MemoryBuffer& x)
	{
		return RectifiedLinearUnitPrimeWorker(z, x, 0.01);
	}

	EXPORT int _InverseSquareRootLinearUnit(MemoryBuffer& z, const MemoryBuffer& x)
	{
		switch (z.mathDomain)
		{
			case MathDomain::Float:
				CUDA_CALL_SINGLE(__InverseSquareRootLinearUnit__<float>, (float*)z.pointer, (float*)x.pointer, z.size);
				break;
			case MathDomain::Double:
				CUDA_CALL_DOUBLE(__InverseSquareRootLinearUnit__<double>, (double*)z.pointer, (double*)x.pointer, z.size);
				break;
			default:
				return CudaKernelException::_NotImplementedException;
		}
		return hipGetLastError();
	}
	
	EXPORT int _InverseSquareRootLinearUnitPrime(MemoryBuffer& z, const MemoryBuffer& x)
	{
		switch (z.mathDomain)
		{
			case MathDomain::Float:
				CUDA_CALL_SINGLE(__InverseSquareRootLinearUnitPrime__<float>, (float*) z.pointer, (float*) x.pointer, z.size);
				break;
			case MathDomain::Double:
				CUDA_CALL_DOUBLE(__InverseSquareRootLinearUnitPrime__<double>, (double*) z.pointer, (double*) x.pointer, z.size);
				break;
			default:
				return CudaKernelException::_NotImplementedException;
		}
		return hipGetLastError();
	}

	EXPORT int _ExponentialLinearUnit(MemoryBuffer& z, const MemoryBuffer& x)
	{
		switch (z.mathDomain)
		{
			case MathDomain::Float:
				CUDA_CALL_SINGLE(__InverseSquareRootLinearUnit__<float>, (float*)z.pointer, (float*)x.pointer, z.size);
				break;
			case MathDomain::Double:
				CUDA_CALL_DOUBLE(__InverseSquareRootLinearUnit__<double>, (double*)z.pointer, (double*)x.pointer, z.size);
				break;
			default:
				return CudaKernelException::_NotImplementedException;
		}
		return hipGetLastError();
	}
	
	EXPORT int _ExponentialLinearUnitPrime(MemoryBuffer& z, const MemoryBuffer& x)
	{
		switch (z.mathDomain)
		{
			case MathDomain::Float:
				CUDA_CALL_SINGLE(__InverseSquareRootLinearUnitPrime__<float>, (float*) z.pointer, (float*) x.pointer, z.size);
				break;
			case MathDomain::Double:
				CUDA_CALL_DOUBLE(__InverseSquareRootLinearUnitPrime__<double>, (double*) z.pointer, (double*) x.pointer, z.size);
				break;
			default:
				return CudaKernelException::_NotImplementedException;
		}
		return hipGetLastError();
	}

	EXPORT int _BentIdentity(MemoryBuffer& z, const MemoryBuffer& x)
	{
		switch (z.mathDomain)
		{
			case MathDomain::Float:
				CUDA_CALL_SINGLE(__BentIdentity__<float>, (float*)z.pointer, (float*)x.pointer, z.size);
				break;
			case MathDomain::Double:
				CUDA_CALL_DOUBLE(__BentIdentity__<double>, (double*)z.pointer, (double*)x.pointer, z.size);
				break;
			default:
				return CudaKernelException::_NotImplementedException;
		}
		return hipGetLastError();
	}
	
	EXPORT int _BentIdentityPrime(MemoryBuffer& z, const MemoryBuffer& x)
	{
		switch (z.mathDomain)
		{
			case MathDomain::Float:
				CUDA_CALL_SINGLE(__BentIdentityPrime__<float>, (float*) z.pointer, (float*) x.pointer, z.size);
				break;
			case MathDomain::Double:
				CUDA_CALL_DOUBLE(__BentIdentityPrime__<double>, (double*) z.pointer, (double*) x.pointer, z.size);
				break;
			default:
				return CudaKernelException::_NotImplementedException;
		}
		return hipGetLastError();
	}

	EXPORT int _SoftMax(MemoryTile& z, const MemoryTile& x, MemoryBuffer& columnWiseSumCache, MemoryBuffer& onesCache)
	{
		switch (z.mathDomain)
		{
			case MathDomain::Float:
				CUDA_CALL_SINGLE(__SoftMax__<float>, (float*)z.pointer, (float*)x.pointer, z.size);
				break;
			case MathDomain::Double:
				CUDA_CALL_DOUBLE(__SoftMax__<double>, (double*)z.pointer, (double*)x.pointer, z.size);
				break;
			default:
				return CudaKernelException::_NotImplementedException;
		}
		
		if (columnWiseSumCache.size != z.nCols)
		{
			if (columnWiseSumCache.pointer != 0)
				_Free(columnWiseSumCache);
			columnWiseSumCache.pointer = 0;
		}
		
		if (columnWiseSumCache.pointer == 0)
		{
			columnWiseSumCache = MemoryBuffer(0, z.nCols, z.memorySpace, z.mathDomain);
			_Alloc(columnWiseSumCache);
		}
		
		int err = _RowWiseSum(columnWiseSumCache, z, onesCache, MatrixOperation::Transpose);
		if (err)
			return err;
		
		err = _Reciprocal(columnWiseSumCache);
		if (err)
			return err;
		
		err = _ScaleColumns(z, columnWiseSumCache);
		if (err)
			return err;
		
		return hipGetLastError();
	}

	EXPORT int _CrossEntropyCostFunctionSigmoid(double& cost, MemoryBuffer& x, const MemoryBuffer& y)
	{
		switch (x.mathDomain)
		{
			case MathDomain::Float:
				CUDA_CALL_SINGLE(__CrossEntropyCostFunctionSigmoid__<float>, (float*)x.pointer, (float*)y.pointer, x.size);
				break;
			case MathDomain::Double:
				CUDA_CALL_DOUBLE(__CrossEntropyCostFunctionSigmoid__<double>, (double*)x.pointer, (double*)y.pointer, x.size);
				break;
			default:
				return CudaKernelException::_NotImplementedException;
		}
		
		// now sum everything together
		return _Sum(cost, x);
	}

	EXPORT int _CrossEntropyCostFunctionSoftMax(double& cost, MemoryBuffer& x, const MemoryBuffer& y)
	{
		switch (x.mathDomain)
		{
			case MathDomain::Float:
				CUDA_CALL_SINGLE(__CrossEntropyCostFunctionSoftMax__<float>, (float*)x.pointer, (float*)y.pointer, x.size);
				break;
			case MathDomain::Double:
				CUDA_CALL_DOUBLE(__CrossEntropyCostFunctionSoftMax__<double>, (double*)x.pointer, (double*)y.pointer, x.size);
				break;
			default:
				return CudaKernelException::_NotImplementedException;
		}
		
		// now sum everything together
		return _Sum(cost, x);
	}
}