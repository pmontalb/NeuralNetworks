#include "hip/hip_runtime.h"
#include <ObjectiveFunctions.cuh>
#include <CubWrappers.cuh>

template <typename T>
DEVICE T __SigmoidWorker__(const T* RESTRICT x, const unsigned i)
{
	return static_cast<T>(1.0) / (static_cast<T>(1.0) + exp(-x[i]));
}

template <typename T>
GLOBAL void __Sigmoid__(T* RESTRICT z, const T* RESTRICT x, const unsigned sz)
{
	CUDA_FUNCTION_PROLOGUE;
	
	CUDA_FOR_LOOP_PROLOGUE
		z[i] = __SigmoidWorker__<T>(x, i);
	CUDA_FOR_LOOP_EPILOGUE
}

template <typename T>
GLOBAL void __SigmoidPrime__<T>(T* RESTRICT z, const T* RESTRICT x, const unsigned sz)
{
	CUDA_FUNCTION_PROLOGUE;
	
	CUDA_FOR_LOOP_PROLOGUE
		const float sigmoidZ = __SigmoidWorker__<T>(x, i);
		z[i] = sigmoidZ * (static_cast<T>(1.0) - sigmoidZ);
	CUDA_FOR_LOOP_EPILOGUE
}

template <typename T>
GLOBAL void __CrossEntropyCostFunction__(T* RESTRICT x, const T* RESTRICT y, const unsigned sz)
{
	CUDA_FUNCTION_PROLOGUE;
	
	CUDA_FOR_LOOP_PROLOGUE
		const T crossEntropy = -y[i] * log(x[i]) - (1.0 - y[i]) * log(1.0 - x[i]);
		if (!isfinite(crossEntropy))
			x[i] = 0.0;
		else
			x[i] = crossEntropy;
	CUDA_FOR_LOOP_EPILOGUE
}

EXTERN_C
{
	EXPORT int _Sigmoid(MemoryBuffer z, const MemoryBuffer x)
	{
		switch (z.mathDomain)
		{
			case MathDomain::Float:
				CUDA_CALL_SINGLE(__Sigmoid__<float>, (float*)z.pointer, (float*)x.pointer, z.size);
				break;
			case MathDomain::Double:
				CUDA_CALL_DOUBLE(__Sigmoid__<double>, (double*)z.pointer, (double*)x.pointer, z.size);
				break;
			default:
				return CudaKernelException::_NotImplementedException;
		}
		return hipGetLastError();
	}
	
	EXPORT int _SigmoidPrime(MemoryBuffer z, const MemoryBuffer x)
	{
		switch (z.mathDomain)
		{
			case MathDomain::Float:
				CUDA_CALL_SINGLE(__SigmoidPrime__<float>, (float*) z.pointer, (float*) x.pointer, z.size);
				break;
			case MathDomain::Double:
				CUDA_CALL_DOUBLE(__SigmoidPrime__<double>, (double*) z.pointer, (double*) x.pointer, z.size);
				break;
			default:
				return CudaKernelException::_NotImplementedException;
		}
		return hipGetLastError();
	}

	EXPORT int _CrossEntropyCostFunction(double& cost, MemoryBuffer x, const MemoryBuffer y)
	{
		switch (x.mathDomain)
		{
			case MathDomain::Float:
				CUDA_CALL_SINGLE(__CrossEntropyCostFunction__<float>, (float*)x.pointer, (float*)y.pointer, x.size);
				break;
			case MathDomain::Double:
				CUDA_CALL_DOUBLE(__CrossEntropyCostFunction__<double>, (double*)x.pointer, (double*)y.pointer, x.size);
				break;
			default:
				return CudaKernelException::_NotImplementedException;
		}
		
		// now sum everything together
		return _Sum(cost, x);
	}
}