#include "hip/hip_runtime.h"
#include <ObjectiveFunctions.cuh>

template <typename T>
DEVICE T __SigmoidWorker__(const T* RESTRICT x, const unsigned i)
{
	return static_cast<T>(1.0) / (static_cast<T>(1.0) + exp(-x[i]));
}

template <typename T>
GLOBAL void __Sigmoid__(T* RESTRICT z, const T* RESTRICT x, const unsigned sz)
{
	CUDA_FUNCTION_PROLOGUE;
	
	CUDA_FOR_LOOP_PROLOGUE
		z[i] = __SigmoidWorker__<T>(x, i);
	CUDA_FOR_LOOP_EPILOGUE
}

template <typename T>
GLOBAL void __SigmoidPrime__<T>(T* RESTRICT z, const T* RESTRICT x, const unsigned sz)
{
	CUDA_FUNCTION_PROLOGUE;
	
	CUDA_FOR_LOOP_PROLOGUE
		const float sigmoidZ = __SigmoidWorker__<T>(x, i);
		z[i] = sigmoidZ * (static_cast<T>(1.0) - sigmoidZ);
	CUDA_FOR_LOOP_EPILOGUE
}

EXTERN_C
{
	EXPORT int _Sigmoid(MemoryBuffer z, const MemoryBuffer x)
	{
		switch (z.mathDomain)
		{
			case MathDomain::Float:
				CUDA_CALL_SINGLE(__Sigmoid__<float>, (float*)z.pointer, (float*)x.pointer, z.size);
				break;
			case MathDomain::Double:
				CUDA_CALL_DOUBLE(__Sigmoid__<double>, (double*)z.pointer, (double*)x.pointer, z.size);
				break;
			default:
				return CudaKernelException::_NotImplementedException;
		}
		return hipGetLastError();
	}
	
	EXPORT int _SigmoidPrime(MemoryBuffer z, const MemoryBuffer x)
	{
		switch (z.mathDomain)
		{
			case MathDomain::Float:
				CUDA_CALL_SINGLE(__SigmoidPrime__<float>, (float*) z.pointer, (float*) x.pointer, z.size);
				break;
			case MathDomain::Double:
				CUDA_CALL_DOUBLE(__SigmoidPrime__<double>, (double*) z.pointer, (double*) x.pointer, z.size);
				break;
			default:
				return CudaKernelException::_NotImplementedException;
		}
		return hipGetLastError();
	}
}